#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <numeric>
#include <limits>
#include <vector>
#include <time.h>

using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true){
	if (code != hipSuccess){
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void testKernel(char* a, int* neig, int* itr, int* seqSize){
	int num_str = 64 * 32;
	int txt_len = 51;
	int seqidx = blockIdx.x * blockDim.x + threadIdx.x + *itr*num_str;
	for (int i = 0; i < *seqSize; i++){		
		int matchChar = 0;
		for (int j = 0; j < txt_len; j++){
			int idx = i*txt_len + j;
			int tgtidx = seqidx*txt_len + j;
			if (a[tgtidx] == a[idx]) matchChar++;
		}
		if ((float)matchChar / txt_len >= 0.97) neig[seqidx]++;
	}
}

void reportCudaDevice(){
	int deviceCount;
	hipDeviceProp_t deviceProp;
	hipGetDeviceCount(&deviceCount);
	hipGetDeviceProperties(&deviceProp,0);
	cout << deviceProp.name << endl;
	cout << deviceProp.maxThreadsPerBlock << endl;
}


int main(int argc, char* argv[]){
	clock_t t1, t2, tc, ti; //For measure computing time.
	t1 = clock();
	unsigned const int num_blocks = 64;
	unsigned const int num_threads = 32;
	int txt_len = 51;
	const int num_str = num_blocks * num_threads;

	dim3 grid(num_blocks, 1, 1);
	dim3 threads(num_threads, 1, 1);

	vector<string> ID;
	vector<string> seq;
	ifstream fp;
	string line;
	float cutoff = 0.97;
	fp.open(argv[1], ios::in);
	if (!fp){
		std::cout << "fail to open " << argv[1] << std::endl;
		exit(EXIT_FAILURE);
	}
	while (getline(fp, line)){
		if (line.at(0) == '>'){
			ID.push_back(line.substr(1));
		}
		else{
			seq.push_back(line);
		}
	}
	fp.close();
	txt_len = seq.at(0).length();
	std::cout << "seq number = " << seq.size() << std::endl;	
	std::cout << "seq length = " << txt_len << std::endl;

	char *a;
	int *neig;
	char *d_a;
	int *d_neig;
	a = (char *)malloc(seq.size()*txt_len*sizeof(char));
	neig = (int *)malloc(seq.size()*sizeof(int));

	for (int i = 0; i < seq.size(); i++){
		neig[i] = 0;
		for (int j = 0; j < txt_len; j++){
			int idx = i*txt_len + j;
			a[idx] = seq.at(i).at(j);
		}
	}

	gpuErrchk(hipMalloc((void**)&d_a, seq.size()*txt_len*sizeof(char)));
	gpuErrchk(hipMemcpy(d_a, a, seq.size()*txt_len*sizeof(char), hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc((void**)&d_neig, seq.size()*sizeof(int)));
	gpuErrchk(hipMemcpy(d_neig, neig, seq.size()*sizeof(int), hipMemcpyHostToDevice));
	int *d_num_str;
	int *d_txt_len;
	int *d_num_threads;
	float *d_cutoff;
	gpuErrchk(hipMalloc((void**)&d_num_str, sizeof(int)));
	gpuErrchk(hipMalloc((void**)&d_txt_len, sizeof(int)));
	gpuErrchk(hipMalloc((void**)&d_num_threads, sizeof(int)));
	gpuErrchk(hipMalloc((void**)&d_cutoff, sizeof(float)));
	gpuErrchk(hipMemcpy(d_cutoff, &cutoff, sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_txt_len, &txt_len, sizeof(int), hipMemcpyHostToDevice));
	int itr = 0;
	int *d_itr;
	hipMalloc((void**)&d_itr, sizeof(int));
	int iterMax = seq.size() / num_str +1;
	int seqSize = seq.size();
	int *d_seqSize;
	hipMalloc( (void**)&d_seqSize, sizeof(int));
	hipMemcpy( d_seqSize, &seqSize, sizeof(int), hipMemcpyHostToDevice);
	tc = clock();	
	for (itr = 0; itr <= iterMax; itr++){
		hipMemcpy(d_itr, &itr, sizeof(int), hipMemcpyHostToDevice);
		testKernel << <grid, threads >> >(d_a, d_neig, d_itr, d_seqSize);
		hipDeviceSynchronize();
		ti = clock();
		float diff((float)ti - (float)tc);
		float seconds = diff / CLOCKS_PER_SEC;
		cout << "itr=" << itr << ",num=" << itr*num_str << "/" << seq.size() << ",time=" << seconds << endl;
	}

	hipMemcpy(d_itr, &itr, sizeof(int), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	printf("Cuda status: %s\n", hipGetErrorString(hipGetLastError()));
	gpuErrchk(hipMemcpy(neig, d_neig, seq.size()*sizeof(int), hipMemcpyDeviceToHost));
	ofstream myfile;
	myfile.open("dbg.csv");

	for (int i = 0; i < seq.size(); i++){
		myfile << ID.at(i) << ',' << neig[i] << endl;
	}
	myfile.close();
	gpuErrchk(hipFree(d_a));
	gpuErrchk(hipFree(d_neig));
	t2 = clock();
	float diff((float)t2 - (float)t1);
	float seconds = diff / CLOCKS_PER_SEC;
	cout << "time=" << seconds << endl;
	return EXIT_SUCCESS;
}
